#include "hip/hip_runtime.h"
/*
 * fuzzing revealed two problems already
 *   - wrong datatype used while backcopying results to host
 *   - dev_c was not set to zero before use
 *   - For very long vectors (N>1000) relative precision goes down to at worst ~1e-4.
 *       -     It might be nice to explore this in more detail systematically.
*/

#define N 2000
#define NB_FUZZES 10000
#define MAX_EPS 1e-3

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "cudaspecs.h"

// The "kernel" to run on the device
__global__ void dot (float *a, float *b, float *c) {
    __shared__ float tmp[MAX_THREADS_PER_BLOCK];
       
    // Element-wise parallel multiplication across threads
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        tmp[threadIdx.x] = a[index] * b[index];
    } else {
        // In the last thread blocks, we might have some
        // empty threads at the ends.
        // Thus, we fill the corresponding vector elements
        // with zero, so that we may safely add up the
        // complete vector later.
        tmp[threadIdx.x] = 0;
    }
    
    // All thread wait here, until all threads reach this line
    __syncthreads();
    
    // Thread 0 sums up the products and writes sum back to *c
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < blockDim.x; i++) {
            sum += tmp[i];
        }
        // Add result of this threadblock to the overall sum
        // Needs to be atomic to avoid race conditions.
        atomicAdd(c, sum);
    }
}


int main() {
    // Memory size per vector
    int size = N * sizeof(float);
    
    // Allocate host memory
    float *a, *b, *c;
    a = (float*) malloc(size);
    b = (float*) malloc(size);
    c = (float*) malloc(sizeof(float));
    
    // Allocate device memory
    // This needs a pointer to a pointer, hence we
    // pass the _address_ of our pointer
    float *dev_a, *dev_b, *dev_c; 
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, sizeof(float));
    
    // Fuzz NB_FUZZES times
    srand(time(NULL));
    for (int f = 0; f < NB_FUZZES; f++) {
        
        // Initialise randomly
        for (int i = 0; i < N; i++){
            a[i] = (float) 100*rand()/(float)(RAND_MAX) - 50.0;
            b[i] = (float) 100*rand()/(float)(RAND_MAX) - 50.0;
        }
        *c = 0;
        
        // Copy to device
        hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_c, c, sizeof(float), hipMemcpyHostToDevice);
        
        // Launch the "kernel"
        int nb_threads_per_block = MAX_THREADS_PER_BLOCK;
        int nb_threadblocks = 1+(N / MAX_THREADS_PER_BLOCK);
        assert (nb_threadblocks <= MAX_GRIDSIZE_1D);
        assert (nb_threads_per_block <= MAX_THREADS_PER_BLOCK);
        dot<<<nb_threadblocks, nb_threads_per_block>>>(dev_a, dev_b, dev_c);
        
        // Copy back the result to c
        hipMemcpy(c, dev_c, sizeof(float), hipMemcpyDeviceToHost);
        
        // Calculate the result on the local machine to compare
        float result = 0;
        for (int i = 0; i < N; i++) {
            result += a[i]*b[i];
        }
        
        printf("Iteration %-5d: Local result %-15f, CUDA result: %-15f, Rel. Difference: %-15f \n", f, result, *c, (*c-result)/result);
        assert (abs(*c-result)/result < MAX_EPS);
    }
    
    
    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    // Done.
    return 0;
}
