#define N 1000
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_GRIDSIZE_1D 2147483647
#define MAX_GRIDSIZE_2D 65535
#define MAX_GRIDSIZE_3D 65535

#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// The "kernel" to run on the device
__global__ void dot (int *a, int *b, int *c) {
    // Memory shared between all threads of a thread block
    __shared__ int tmp[N];
    
    // Element-wise parallel multiplication across threads
    tmp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
    
    // All thread wait here, until all threads reach this line
    __syncthreads();
    
    // Thread 0 sums up the products and writes sum back to *c
    if (threadIdx.x == 0) {
        int sum = 0;
        for (int i = 0; i < N; i++) {
            sum += tmp[i];
        }
        *c = sum;
    }
}


int main() {
    // Memory size per vector
    int size = N * sizeof(int);
    
    // Allocate host memory
    int *a, *b, *c;
    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(sizeof(int));
    
    // Initialise randomly
    srand(time(NULL));
    for (int i = 0; i < N; i++){
        a[i] = 2; //rand() % 1000;
        b[i] = 3; //rand() % 1000;
    }
    
    // Allocate device memory
    // This needs a pointer to a pointer, hence we
    // pass the _address_ of our pointer
    int *dev_a, *dev_b, *dev_c; 
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, sizeof(int));
    
    // Copy to device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    
    // Launch the "kernel"
    int nb_threadblocks = 1;
    int nb_threads_per_block = N;
    assert (nb_threadblocks < MAX_GRIDSIZE_1D);
    assert (nb_threads_per_block < MAX_THREADS_PER_BLOCK);
    dot<<<nb_threadblocks, nb_threads_per_block>>>(dev_a, dev_b, dev_c);
    
    // Copy back
    hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    // Check
    printf("%i should be %i\n", *c, N*6);
    
    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    // Done.
    return 0;
}
