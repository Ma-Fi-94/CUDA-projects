#include "hip/hip_runtime.h"
#define N (100*1024+5)

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "cudaspecs.h"

// The "kernel" to run on the device
__global__ void dot (float *a, float *b, float *c) {
    __shared__ float tmp[MAX_THREADS_PER_BLOCK];
       
    // Element-wise parallel multiplication across threads
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        tmp[threadIdx.x] = a[index] * b[index];
    } else {
        // In the last thread blocks, we might have some
        // empty threads at the ends.
        // Thus, we fill the corresponding vector elements
        // with zero, so that we may safely add up the
        // complete vector later.
        tmp[threadIdx.x] = 0;
    }
    
    // All thread wait here, until all threads reach this line
    __syncthreads();
    
    // Thread 0 sums up the products and writes sum back to *c
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < blockDim.x; i++) {
            sum += tmp[i];
        }
        // Add result of this threadblock to the overall sum
        // Needs to be atomic to avoid race conditions.
        atomicAdd(c, sum);
    }
}


int main() {
    // Memory size per vector
    int size = N * sizeof(float);
    
    // Allocate host memory
    float *a, *b, *c;
    a = (float*) malloc(size);
    b = (float*) malloc(size);
    c = (float*) malloc(sizeof(float));
    *c = 0;
    
    // Initialise randomly
    srand(time(NULL));
    for (int i = 0; i < N; i++){
        a[i] = 2; //rand() % 1000;
        b[i] = 3; //rand() % 1000;
    }
    
    // Allocate device memory
    // This needs a pointer to a pointer, hence we
    // pass the _address_ of our pointer
    float *dev_a, *dev_b, *dev_c; 
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, sizeof(float));
    
    // Copy to device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);
    
    // Launch the "kernel"
    int nb_threads_per_block = MAX_THREADS_PER_BLOCK;
    int nb_threadblocks = 1+(N / MAX_THREADS_PER_BLOCK);
    printf("N=%i, thus %i blocks a %i threads.\n", N, nb_threadblocks, nb_threads_per_block);
    printf("Last thread only computes %i elements\n", N % MAX_THREADS_PER_BLOCK);
    assert (nb_threadblocks <= MAX_GRIDSIZE_1D);
    assert (nb_threads_per_block <= MAX_THREADS_PER_BLOCK);
    dot<<<nb_threadblocks, nb_threads_per_block>>>(dev_a, dev_b, dev_c);
    
    // Copy back
    hipMemcpy(c, dev_c, sizeof(float), hipMemcpyDeviceToHost);
    
    // The result
    printf("Result: %f, should be %i\n", *c, N*6);
    assert (*c == N*6);

    
    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    // Done.
    return 0;
}
