// From deviceQuery
// TODO: make header file for this
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_GRIDSIZE_1D 2147483647
#define MAX_GRIDSIZE_2D 65535
#define MAX_GRIDSIZE_3D 65535

// Simulation parameters
#define MAXSTEP 100


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>


// The "kernel" to run on the device
__global__ void propagate(int *lattice, int* lattice_new, int XSIZE, int YSIZE) {
    int i = blockIdx.x;
    lattice_new[i] = lattice[i];
    
    int X = i / YSIZE;
    int Y = i % YSIZE;

    if (X > 0 & Y > 0 & X < XSIZE-1 & Y < YSIZE-1) {
        int n = lattice[i-YSIZE-1] + lattice[i-YSIZE] + lattice[i-YSIZE+1] +
                lattice[i-1] + lattice[i+1] + 
                lattice[i+YSIZE-1] + lattice[i+YSIZE] + lattice[i+YSIZE+1];
        
        if (lattice[i] == 0 & n == 3) {
            lattice_new[i] = 1;
            return;
        }
        
        if (lattice[i] == 1 & n < 2) {
            lattice_new[i] = 0;
            return;
        }

        if (lattice[i] == 1 & n > 3) {
            lattice_new[i] = 0;
            return;
        }
        return;
    }
    
    return;
}


// Another "kernel" to run on the device
__global__ void update(int *lattice, int* lattice_new) {
    int i = blockIdx.x;
    lattice[i] = lattice_new[i];
}


int main(int argc, char *argv[]) {
    assert(argc==3);   
    int XSIZE = atoi(argv[1]);
    int YSIZE = atoi(argv[2]);
    
    // Allocate host memory
    int *lattice;
    lattice = (int*) calloc(XSIZE * YSIZE, sizeof(int));
      
    // Initialise host array
    srand(time(NULL));
    for (int i = 1; i < XSIZE-1; i++) {
        for (int j = 1; j < YSIZE-1; j++) {
            lattice[i*YSIZE+j] = rand() % 100 < 50;
        }
    }
    
    // Start timer
    clock_t begin = clock();
    
    // Allocate device memory
    // This needs a pointer to a pointer, hence we
    // pass the _address_ of our pointer
    int *d_lattice, *d_lattice_new; 
    hipMalloc((void**) &d_lattice, sizeof(int) * XSIZE * YSIZE);
    hipMalloc((void**) &d_lattice_new, sizeof(int) * XSIZE * YSIZE);
    
    
    // Copy lattice from host to device
    hipMemcpy(d_lattice, lattice, sizeof(int) * XSIZE * YSIZE, hipMemcpyHostToDevice);

    // Preparation and sanity checks for kernel launches
    int T = 1;  // nb. threads per thread block
    int G = XSIZE*YSIZE;  // nb. thread blocks
    assert (T <= MAX_THREADS_PER_BLOCK);
    assert (G <= MAX_GRIDSIZE_1D);
    
    // Main simulation loop
    for (int i = 0; i <= MAXSTEP; i++) {               
        propagate<<<G,T>>>(d_lattice, d_lattice_new, XSIZE, YSIZE);
        update<<<G,T>>>(d_lattice, d_lattice_new);
    }
      
   
    // Deallocate device memory
    hipFree(d_lattice);
   
    // Stop timer
    clock_t end = clock();
    
    printf("GPU,%i,%li\n", XSIZE, end-begin);
   
    // Deallocate host memory
    free(lattice);
}

